#include "hip/hip_runtime.h"
#include "device_operations.h"
#include <assert.h>
#include <stdlib.h>
#include <hip/hip_runtime_api.h>	
#include <chrono>

__global__ void gpuMatrixConv3D(float* image, float* mask, float* weight, float* result, int imageRows, int imageCols, int maskRC, int maskDepth, int resultRows, int resultCols, float* bias, float* mean, float* variance, int strideRows, int strideCols) {
    
    __shared__ float sharedMask[3*3*3];

    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int channel = blockIdx.z;
    int sharedIdx = 0, maskIdx = 0, imgIdx = 0 ;

    if (row < resultRows && col < resultCols) {
         
         //Load mask into shared memory
        if (threadIdx.x < maskRC && threadIdx.y < maskRC){
            for(int d = 0 ; d < maskDepth; d++ ){
                sharedIdx = d*maskRC*maskRC + threadIdx.y * maskRC + threadIdx.x;
                maskIdx =  channel *maskRC*maskRC*maskDepth + d*maskRC*maskRC + threadIdx.y * maskRC + threadIdx.x;
                sharedMask[sharedIdx] = mask[maskIdx];
            }
        } 
        
        // Synchronize threads to ensure all data is loaded into shared memory
        __syncthreads();
        int imageRowsCols = imageRows * imageCols;
        float sum = 0.0;

        // Convolution operation
        for (int maskRow = 0; maskRow < maskRC; maskRow++) {
            for (int maskCol = 0; maskCol < maskRC; maskCol++) {
                for (int dep = 0; dep < maskDepth; dep++) {
                    imgIdx = (row * strideRows + maskRow) * imageCols + col * strideCols + maskCol + dep * imageRowsCols;
                    maskIdx  = dep * maskRC*maskRC + maskRow * maskRC + maskCol;
                    sum += image[imgIdx]*sharedMask[maskIdx];
                }   
            }
        }

        // Batch normalization
        float normalized_sum = ((sum - mean[channel]) / (sqrtf(variance[channel]) + 0.00001))*weight[channel] + bias[channel];

        // ReLU6 activation
        float relu6_output = fminf(fmaxf(normalized_sum, 0.0f), 6.0f);

        // Store the result
        result[channel*resultCols*resultRows + row * resultCols + col] = relu6_output;
    }
}


int main() {
    // Dimension declaration and definition
    int imgRow, imgCol, imgChannels,kernel_dims,padding,output_channels,stride;
    kernel_dims = 3;
    output_channels = 16;
    padding = 1;
    imgChannels = 3;
    stride = 2;
    
    // model allocation in HOST (cpu)
    float *kernel = (float*) malloc(sizeof(float) * output_channels * kernel_dims * kernel_dims * kernel_dims);
    float *bias = new float[output_channels];
    float *means = new float[output_channels];
    float *variances = new float[output_channels];
    float *weights = new float[output_channels];

    // model loading in HOST (cpu). Same whatever the image is.
    loadKernels("./model/stem_params/0.weight.txt", kernel, kernel_dims, output_channels);
    loadBatchParams("./model/stem_params/1.weight.txt",weights,output_channels);
    loadBatchParams("./model/stem_params/1.bias.txt", bias, output_channels);
    loadBatchParams("./model/stem_params/1.running_mean.txt", means, output_channels);
    loadBatchParams("./model/stem_params/1.running_var.txt", variances, output_channels);


    // loading image...
    float* image = loadImage("./images_processed/imgtest.txt" ,&imgRow, &imgCol, imgChannels);
    
    // computing outputRow and outputCol and then space allocation to store feature maps.
    int outputRow = (imgRow + 2*padding - kernel_dims)/stride +1;
    int outputCol = (imgCol +2*padding - kernel_dims)/stride +1;
    float *output = (float*)malloc(sizeof(float) * output_channels * outputRow * outputCol);

    // padding
    auto startPadding = std::chrono::high_resolution_clock::now();
    image = imgPadding(image, imgRow, imgCol, imgChannels, padding);
    auto endPadding = std::chrono::high_resolution_clock::now();
    
    // Calcola la durata dell'operazione di padding
    std::chrono::duration<float> durationPadding = endPadding - startPadding;
    float paddingDuration_ms = durationPadding.count()*1000;

    // Final allocation check
    if (image == NULL || kernel == NULL || output == NULL) {
        std::cerr << "Allocation (host) error." << std::endl;
        exit(EXIT_FAILURE);
    }

    // declaring device pointers.
    float *d_image, *d_output, *d_kernel, *d_bias, *d_means, *d_variances, *d_weights;

    // allocating space on the device
    hipMalloc((void**)&d_image, sizeof(float) * imgChannels * (imgRow + 2*padding) * (imgCol + 2*padding));
    hipMalloc((void**)&d_kernel, sizeof(float) * output_channels * kernel_dims * kernel_dims * kernel_dims);
    hipMalloc((void**)&d_output, sizeof(float) * output_channels * outputRow * outputCol); 
    hipMalloc((void**)&d_bias, sizeof(float) * output_channels);
    hipMalloc((void**)&d_means, sizeof(float) * output_channels);
    hipMalloc((void**)&d_variances, sizeof(float) * output_channels);
    hipMalloc((void**)&d_weights, sizeof(float) * output_channels);
    
    // shift
    hipMemcpy(d_image, image, sizeof(float) * imgChannels * (imgRow + 2*padding) * (imgCol + 2*padding), hipMemcpyHostToDevice);
    hipMemcpy(d_kernel, kernel, sizeof(float) * output_channels * kernel_dims * kernel_dims * kernel_dims, hipMemcpyHostToDevice);
    hipMemcpy(d_bias, bias, sizeof(float) * output_channels, hipMemcpyHostToDevice);
    hipMemcpy(d_means, means, sizeof(float) * output_channels, hipMemcpyHostToDevice);
    hipMemcpy(d_variances, variances, sizeof(float) * output_channels, hipMemcpyHostToDevice);
    hipMemcpy(d_weights, weights, sizeof(float) * output_channels, hipMemcpyHostToDevice);

	//grid setup.
	int threadsPerBlock = 16;
	int gridCols = ceil(float(outputCol) / float(threadsPerBlock));
	int gridRows = ceil(float(outputRow) / float(threadsPerBlock));
    int gridChannels = output_channels;
	dim3 gridDim(gridCols, gridRows,gridChannels);
	dim3 blockDim(threadsPerBlock, threadsPerBlock);

    // starting convolution (paralel,gpu)
	gpuMatrixConv3D << < gridDim, blockDim >> > (d_image, d_kernel, d_weights, d_output, imgRow + 2*padding, imgCol + 2*padding, imgChannels, kernel_dims, outputRow, outputCol,d_bias,d_means,d_variances,stride,stride);

    // waiting cuda get the job done to store.
    hipDeviceSynchronize();

    // Copy the result back to host
    hipMemcpy(output, d_output, sizeof(float) * output_channels * outputRow * outputCol, hipMemcpyDeviceToHost);
    // store feature map
    storeConvolution("./test_output/convolution_results/stem_conv_mask_shared.txt", output, outputRow, outputCol, output_channels);

    hipFree(d_image);
    hipFree(d_output);
    hipFree(d_kernel);
    hipFree(d_weights);
    hipFree(d_bias);
    hipFree(d_means);
    hipFree(d_variances);
    return 0;
}