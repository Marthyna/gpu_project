#include <hip/hip_runtime.h>
#include <iostream>

int main() {
    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    for (int i = 0; i < deviceCount; ++i) {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        std::cout << "Device " << i << ": " << prop.name << std::endl;
        std::cout << "    Shared Memory Per Block: " << prop.sharedMemPerBlock << " bytes" << std::endl;
        std::cout << "    Streaming Multiprocessors: " << prop.multiProcessorCount << std::endl;
    }
    return 0;
}
